#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
__global__ void hello_from_gpu()

{
	const int b =blockIdx.x;
	const int tx=threadIdx.x;
	const int ty=threadIdx.y;

        //cout<<b<<"\n"<<endl;
	printf("%d%d%d\n",b,tx,ty);

}


int main(void)
{
  const dim3 block_size(2,4);
  hello_from_gpu<<<2,block_size>>>();
  hipDeviceSynchronize();
  //cudaDeviceSynchronize();

  return 0;
}

