
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
using namespace std;

__global__ void hello_from_gpu()
{
    const int b = blockIdx.x;
    const int c = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
   // cout<<b<<endl;
    printf("Hello World from block-(%d,%d) and thread-(%d, %d)!\n", b,c, tx, ty);
}

int main(void)
{
    const dim3 grid(3, 2);
    const dim3 block(5, 3);
    //const dim3 block_size(2, 4);
    hello_from_gpu<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}

